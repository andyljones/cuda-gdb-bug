#include "hip/hip_runtime.h"
#include <ATen/ATen.h> 

__global__ void increment(float* x) {
    x[threadIdx.x] = x[threadIdx.x] + 1;
}

__host__ int main(int argc, char** argv) {
    auto x = at::ones({1}, at::device(at::kCUDA));

    increment<<<1, 1>>>((float*) x.data_ptr());

    hipDeviceSynchronize();
    printf("(bad) x[0]: %f\n", x[0].item<float>());
    return 0;
} 
