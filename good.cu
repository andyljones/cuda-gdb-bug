
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void increment(float* x) {
    x[threadIdx.x] = x[threadIdx.x] + 1;
}

__host__ int main(int argc, char** argv) {
    float *x;
    hipMallocManaged(&x, 1*sizeof(float));
    x[0] = 1.f;

    increment<<<1, 1>>>(x);

    hipDeviceSynchronize();
    printf("(good) x[0]: %f\n", x[0]);

    hipFree(x);
    return 0;
}